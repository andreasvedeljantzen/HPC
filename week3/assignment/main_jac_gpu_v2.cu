#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "func.h"
#include <omp.h>

// setting GPU device
const int device = 0;
#define BLOCK_SIZE 16


__global__ void jac_mp_v3(int N, double delta, int max_iter, double *f, double *d_u, double *d_u_old) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	// Update u
	d_u[i*N + j] = 0.25 * (d_u_old[(i-1)*N + j] + d_u_old[(i+1)*N + j] + d_u_old[i*N + (j-1)] + d_u_old[i*N + (j+1)] + delta*delta*f[i*N + j]);
			}
		}
	}

int main(int argc, char *argv[]) {

    //setting up device
    hipSetDevice(device);

    // timing
    //double ts, te, mflops, memory,flop;
    int max_iter, loops, N;
        
    // command line argument sets the dimensions of the image
    if (argc == 4 ) {
        N = atoi(argv[1]) + 2;
        max_iter = atoi(argv[2]);
    }
    else {
        // use default N
        N = 32 + 2;
        max_iter = 100;
    }

    // arrays
    double   *d_f, *d_u, *d_u_old;
    double   *h_f, *h_u, *h_u_old;
    int size_f = sizeof(double)*N*N;
    int size_u = sizeof(double)*N*N;
    int size_u_old = sizeof(double)*N*N;

    // GPU
    // Allocate memory on host and device
    hipMalloc((void**)&d_f, size_f);
    hipMalloc((void**)&d_u, size_u);
    hipMalloc((void**)&d_u_old, size_u_old);
    //h_f = (double*)malloc(size_f);
    hipHostMalloc((void**)&h_f, size_f);
    hipHostMalloc((void**)&h_u, size_u);
    hipHostMalloc((void**)&h_u_old, size_u_old);
    
    if (d_f == NULL || d_u == NULL || d_u_old ==NULL) {
       fprintf(stderr, "memory allocation failed!\n");
       return(1);
    }
    if (h_f == NULL || h_u == NULL || h_u_old ==NULL) {
       fprintf(stderr, "memory allocation failed!\n");
       return(1);
    }
    double time, time_end, time_IO_1, time_IO_2, time_compute, time_compute_end,tot_time_compute;	
    time = omp_get_wtime();

    double delta = 2.0/N;

    int i,j;
    for (i = 0; i < N; i++){
        for (j = 0; j < N; j++){
            if (i >= N * 0.5  &&  i <= N * 2.0/3.0  &&  j >= N * 1.0/6.0  &&  j <= N * 1.0/3.0)
                h_f[i*N + j] = 200.0;
            else
                h_f[i*N + j] = 0.0; 

            if (i == (N - 1) || i == 0 || j == (N - 1)){
                h_u[i*N + j] = 20.0;
                h_u_old[i*N + j] = 20.0;
            }
            else{
                h_u[i*N + j] = 0.0;
                h_u_old[i*N + j] = 0.0;
            } 
        }
    }
    
    hipMemcpy(d_f, h_f, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_u, h_u, size_u, hipMemcpyHostToDevice);
    hipMemcpy(d_u_old, h_u_old, size_u_old, hipMemcpyHostToDevice);
    time_IO_1 = omp_get_wtime()- time;

    dim3 dimGrid(512,8,1); // 4096 blocks in total
    dim3 dimBlock(16,16,1);// 256 threads per block
    
    // do program
    //ts = omp_get_wtime();
    int k;
    k=0;
    //double *temp;
    time_compute = omp_get_wtime();
    while (k < max_iter) {
	//Set u_old = u
	//temp = h_u;
	//h_u = h_u_old;
	//h_u_old = temp;
	hipMemcpy(d_u, h_u, size_u, hipMemcpyHostToDevice);
        hipMemcpy(d_u_old, h_u_old, size_u_old, hipMemcpyHostToDevice);
        jac_mp_v3<<<dimGrid, dimBlock>>>(N, delta, max_iter,d_f,d_u,d_u_old);
	hipMemcpy(h_u_old, d_u, size_u_old, hipMemcpyDeviceToHost);
	hipMemcpy(h_u, d_u_old, size_u, hipMemcpyDeviceToHost);
	k++;
	}

    hipDeviceSynchronize();
    time_compute_end = omp_get_wtime();
    // end program

    // Copy result back to host
    hipMemcpy(h_u_old, d_u_old, size_u_old, hipMemcpyDeviceToHost);
    hipMemcpy(h_u, d_u, size_u, hipMemcpyDeviceToHost);

    //timing
    time_end = omp_get_wtime();
    time_IO_2 = time_end - time_compute_end;
    tot_time_compute = time_compute_end - time_compute;

	//print to see wheter it is right   
    for (int i = 0; i < N; i++) {
	for (int j = 0; j < N; j++) {
	    printf("%g\t", h_u[i*N+j]);
			}
	printf("\n");
	}

    //flops
    //flop=max_iter * (double)(N-2) * (double)(N-2) * 10.0;

    // calculate mflops in O
    //mflops  = flop * 1.0e-06 * loops / te;
    //memory  = 3.0 * (double)(N-2) * (double)(N-2) * sizeof(double);
    
    //printf("%d\t", n_cores);
    //printf("%g\t", memory);
    //printf("%g\t", mflops);
    //printf("%g\n", te / loops);

    // stats
    double GB = 1.0e-09;
    double gflops  = (N * N * 2 / tot_time_compute) * GB;
    double memory  = size_f + size_u + size_u_old;
    double memoryGBs  = memory * GB * (1 / tot_time_compute);

    printf("%g\t", memory); // footprint
    printf("%g\t", gflops); // Gflops
    printf("%g\t", gflops / 70.65); // pct. Gflops

    printf("%g\t", memoryGBs); // bandwidth GB/s
    printf("%g\t", memoryGBs / 8.98); // pct. bandwidth GB/s

    printf("%g\t", time_end - time); // total time
    printf("%g\t", time_IO_1 + time_IO_2); // I/O time
    printf("%g\n", tot_time_compute); // compute time

    // Cleanup
    hipHostFree(h_f);
    hipHostFree(h_u);
    hipHostFree(h_u_old); 
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_u_old);
 
    // end program
    return(0);
}

