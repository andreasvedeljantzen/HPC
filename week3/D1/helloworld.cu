#include "hip/hip_runtime.h"
int main(int argc, char **argv)
{
	// Variable tid is in local memory and private to each thread
 	int tid;

	// Transfer data from host to device
	hipMemcpy(...);

	// Kernel lauch
	kernel<<<4, 64>>>();
	__global__ void kernel() {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		printf("Hello world! Im thread %ithreadIdx.x out of %iblockIdx.x . My Global thread id is %ithreadIdx out of %iGridDim.x");
	};

	hipDeviceSynchronize();

	// Transfer results from device to host
	hipMemcpy(...);

 	// Built-in variables like threadIdx.x are in local memory

} 