#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>


__global__ void kernel() {
		i = blockIdx.x * blockDim.x + threadIdx.x;
		printf("Hello world! Im thread %i out of %i . My Global thread id is %i out of %i \n", threadIdx.x, blockIdx.x, i, gridDim.x*blockDim.x );
	};

int main(int argc, char **argv)
{
	const int device = 0;
	hipSetDevice(device); 

	// Kernel lauch

	kernel<<<4, 64>>>();

	hipDeviceSynchronize();

	return(0);

};

